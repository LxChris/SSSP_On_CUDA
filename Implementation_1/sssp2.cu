#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "./utilities/timer.hpp"
#include "./utilities/graph.hpp"
#include "./utilities/gpu_error_check.cuh"

/*

Basic implementation

*/

int numNodes;
int numEdges;

int* dist;
int* previousNode;
int ** graph;
bool* finished;
int* graph_static;

void init(Graph* graphData, int source) {
    numNodes = graphData->numNodes;
    graph = graphData->graph;

    int size = numNodes;

    dist = new int[size];
    previousNode = new int[size];
    finished = new bool[size];

    for (int i = 0; i < size; i++) {
        // set the distance to the source node
        dist[i] = graph[source][i];
        finished[i] = false;
        if (dist[i] == MAX_DIST) {
            previousNode[i] = -1;
        }
        else {
            previousNode[i] = source;
        }
    }
    // set dist[source] = 0
    dist[source] = 0;

}

void printShortestDistance(int source) {
    int diameter = 0;
    for (int i = 0; i < numNodes; i++) {
        if(dist[i] != MAX_DIST){
            if(dist[i] > diameter){
                diameter = dist[i];
            }
            // printf("Shortest distance from node: %d to source: %d: is: %d\n", i, source, dist[i]);
        }else{
            // printf("Shortest distance from node: %d to source: %d: is: INF\n", i, source);
        } 
    }
    printf("Diameter: %d\n", diameter);
}

void printGraph(){
    printf("\n\nGraph");
    for (int i = 0; i < numNodes; i++){
        for (int j = 0; j < numNodes; j++){
            printf("%d ", graph[i][j]);
        }
        printf("\n");
    }
}

void printFinished(){
    printf("Finished array\n");
    for(int i = 0; i < numNodes; i++){
        printf("Node: %d, status: %d\n", i, finished[i]);
    }
    printf("\n");
}

void imcompletedAndConnectedNode(){
    int count = 0;
    for(int i = 0; i < numNodes; i++){
        if((!finished[i] && (dist[i] != MAX_DIST))){
            count++;
        }
    }
    printf("Imcompleted Nodes Number: %d\n", count);
}

void dijkstraOnCPU(int source) {
    Timer timer;
    int size = numNodes;
    int numIteration = 0;
    
    // Find the connected nodes to the source point
    // Set the source point
    dist[source] = 0;
    finished[source] = true;

    timer.start();
    for (int i = 0; i < size; i++) {
        int mindist = MAX_DIST;
        // U is the closet point to source, u is not finished yet
        int u = source;

        numIteration++;

        for (int j = 0; j < size; j++) {
            if ((j != u) && (!finished[j]) && dist[j] < mindist) {
                u = j;
                mindist = dist[j];
            }
        }
        if (u == source){
            // Completed
            break;
        }

        finished[u] = true;

        for (int j = 0; j < size; j++) {
            if ((j != u) && (!finished[j]) && graph[u][j] < MAX_DIST) {
                // Find the shorter path
                if (dist[u] + graph[u][j] < dist[j]) {
                    // Update dist
                    dist[j] = dist[u] + graph[u][j];
                    // Update its previous point
                    previousNode[j] = u;
                }
            }
        }
    }
    printf("Number of Iteration Executed: %d\n", numIteration);
    printf("The execution time of SSSP on CPU: %d ms\n", timer.stop());
    // printShortestDistance(0);
}

__global__ void dijkstraOnGPU_kernel1(int numNodes, 
                                int sourceId,
                                int partSize,
                                int* graphData,
                                bool* finished,
                                int* dist,
                                int* prev,
                                int* closestNodeId,
                                int* minimumDist,
                                bool* completed) {
    // kernel to compute the closest node 
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startNodeId = threadId * partSize;
    int endNodeId = (threadId + 1) * partSize;
    if(endNodeId > numNodes){
        endNodeId = numNodes;
    } 

    if(startNodeId > numNodes) return; 

    // printf("Thread: %d process data from: %d to %d \n", threadId, startNodeId, endNodeId);


    for(int nodeId = startNodeId; nodeId < endNodeId; nodeId++){
        if (!finished[nodeId] && dist[nodeId] < *minimumDist){
            *closestNodeId = nodeId;
            *minimumDist = dist[nodeId];
            *completed = false;
        }
    }

}

__global__ void dijkstraOnGPU_kernel2(int numNodes, 
                                        int sourceId,
                                        int partSize,
                                        int* graphData,
                                        bool* finished,
                                        int* dist,
                                        int* prev,
                                        int* closestNodeId,
                                        int GRAPH_MAX_SIZE) {

    // Based on closest node then update its connected node
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startNodeId = threadId * partSize;
    int endNodeId = (threadId + 1) * partSize;
    if(endNodeId > numNodes){
        endNodeId = numNodes;
    } 
    // int nodeId = threadId;

    if(startNodeId > numNodes) return;

    for(int nodeId = startNodeId; nodeId < endNodeId; nodeId++){
        // Convert to 1-D index
        int index = *closestNodeId * GRAPH_MAX_SIZE + nodeId;
        finished[*closestNodeId] = true;
        // printf("graphData[%d][%d]: is %d \n", *closestNodeId, nodeId, graphData[*closestNodeId][nodeId]);
        if (!finished[nodeId] && graphData[index] < MAX_DIST){
            // Find the shorter path
            if(dist[*closestNodeId] + graphData[index] < dist[nodeId]){
                // Update dist
                dist[nodeId] = dist[*closestNodeId] + graphData[index];
                // Update its previous point
                prev[nodeId] = *closestNodeId;
            }
        }
    }
}

void dijkstraOnGPU(int source){
    Timer timer;
    hipFree(0);
    // Define CPU vars
    int closestNodeId = 6;
    // Define GPU vars
    int* d_graph;   // 2D array is converted to 1-D, row = i / cols, col = i % cols;
    int* d_dist;
    int* d_prev;
    bool* d_finished;
    int* d_closestNodeId;
    int* d_minimumDist;
    bool* d_completed;


    
    gpuErrorcheck(hipMalloc((void **)&d_graph, GRAPH_MAX_SIZE * GRAPH_MAX_SIZE * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_dist, numNodes * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_prev, numNodes * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_finished, numNodes * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_closestNodeId, sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_completed, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_minimumDist, sizeof(int)));

    gpuErrorcheck(hipMemcpy(d_graph, graph[0], GRAPH_MAX_SIZE * GRAPH_MAX_SIZE * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_dist, dist, numNodes * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_prev, previousNode, numNodes * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_finished, finished, numNodes * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_closestNodeId, &closestNodeId, sizeof(int), hipMemcpyHostToDevice));

    bool completed = true;
    int minimumDist = MAX_DIST;
    int numIteration = 0;

    
    // Each block has 128 threads
   
    int numNodesPerPart = 32;
    int numThreadPerBlock = 64;
    int numBlock = (numNodes) / (numNodesPerPart * numThreadPerBlock) + 1;
    
    timer.start();
    do{
        numIteration++;
        completed = true;
        minimumDist = MAX_DIST;

        gpuErrorcheck(hipMemcpy(d_completed, &completed, sizeof(bool), hipMemcpyHostToDevice));
        gpuErrorcheck(hipMemcpy(d_minimumDist, &minimumDist, sizeof(int), hipMemcpyHostToDevice));

        if(numIteration % 2 == 1){
            dijkstraOnGPU_kernel1<<< numBlock, numThreadPerBlock >>>(numNodes,
                source,
                numNodesPerPart,
                d_graph,
                d_finished,
                d_dist,
                d_prev,
                d_closestNodeId,
                d_minimumDist,
                d_completed);
            gpuErrorcheck(hipMemcpy(&completed, d_completed, sizeof(bool), hipMemcpyDeviceToHost));
        }else{
            dijkstraOnGPU_kernel2<<<numBlock, numThreadPerBlock>>>(numNodes,
                source,
                numNodesPerPart,
                d_graph,
                d_finished,
                d_dist,
                d_prev,
                d_closestNodeId,
                GRAPH_MAX_SIZE);
            completed = false;
        }

        
        gpuErrorcheck(hipPeekAtLastError());
        gpuErrorcheck(hipDeviceSynchronize());  
    }while(!completed);

    printf("Number of Iteration Executed: %d\n", numIteration);
    printf("The execution time of SSSP on GPU: %d ms\n", timer.stop());
    hipMemcpy(&closestNodeId, d_closestNodeId, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(dist, d_dist, numNodes * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_graph);
    hipFree(d_dist);
    hipFree(d_prev);
    hipFree(d_finished);
    hipFree(d_closestNodeId);
    hipFree(d_minimumDist);
    hipFree(d_completed);
}


int main() {

    // Graph graph1("datasets/simpleGragh1.txt");
    // Graph graph1("datasets/email-Eu-core-SIMPLE.txt");
    Graph graph1("datasets/email-Eu-core.txt");
    // Graph graph1("datasets/Wiki-Vote.txt");
    // Graph graph1("datasets/simpleGragh2.txt");
    // Graph graph1("datasets/CA-GrQc.txt");
     //Graph graph("datasets/testGraph.txt");
    graph1.readGraph();
    int sourceId = 0;

    init(&graph1, sourceId);   // source 0
        
    // Run SSSP on CPU
    dijkstraOnCPU(sourceId);
    printShortestDistance(sourceId);

    init(&graph1, sourceId);   // source 0

    // Run SSSP on GPU
    dijkstraOnGPU(sourceId);
    printShortestDistance(sourceId);

    return 0;
}