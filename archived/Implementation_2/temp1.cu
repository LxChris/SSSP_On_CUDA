#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "./utilities/timer.hpp"
#include "./utilities/graph.hpp"
#include "./utilities/gpu_error_check.cuh"
#include "./utilities/global.hpp"
#include "./utilities/argument_parser.hpp"
#include <omp.h>


uint* sssp_CPU_parallel(Graph *graph, int source) {
    int numNodes = graph->numNodes;
    int numEdges = graph->numEdges;
    uint *dist = new uint[numNodes];
    uint *preNode = new uint[numNodes];
    bool *processed = new bool[numNodes];
    uint *edgesSource = new uint[numEdges];
    uint *edgesEnd = new uint[numEdges];
    uint *edgesWeight = new uint[numEdges];

    for (int i = 0; i < numNodes; i++) {
        dist[i] = MAX_DIST;
        preNode[i] = uint(-1);
        processed[i] = false;
    }

    for (int i = 0; i < numEdges;i ++) {
        Edge edge = graph->edges.at(i);
        edgesSource[i] = edge.source;
        edgesEnd[i] = edge.end;
        edgesWeight[i] = edge.weight;

        if (edge.source == source) {
            if (edge.weight < dist[edge.end]) {
                dist[edge.end] = edge.weight;
                preNode[edge.end] = source;
            }
        } else {
            // Case: edge.source != source
            continue;
        }
    }
    
    Timer timer;
    bool finished = false;
    uint numIteration = 0;

    dist[source] = 0;
    preNode[source] = 0;
    processed[source] = true;

    timer.start();
    while(!finished) {
        finished = true;
        numIteration++;
        
        #pragma omp parallel 
        {   
            // #pragma omp master 
            int threadId = omp_get_thread_num();
            int numThreads = omp_get_num_threads();
            int numEdgesPerThread = numEdges / numThreads + 1;
            int start = threadId * numEdgesPerThread;
            int end = (threadId + 1) * numEdgesPerThread;
            if (start > numEdges) {
                start = numEdges;
            }
            
            if (end > numEdges) {
                end = numEdges;
            }

            for (int i = start; i < end; i++) {
                uint source = edgesSource[i];
                uint end = edgesEnd[i];
                uint weight = edgesWeight[i];

                if (dist[source] + weight < dist[end]) {
                    // #pragma omp atomic
                    dist[end] = dist[source] + weight;
                    // #pragma omp atomic
                    preNode[end] = source;
                    finished = false;
                }
            }
        }
    }
    timer.stop();

    printf("Process Done!\n");
    printf("Number of Iteration: %d\n", numIteration);
    printf("The execution time of SSSP on CPU(OpenMP): %f ms\n", timer.elapsedTime());

    return dist;

}

__global__ void sssp_GPU_Kernel(int numEdges,
                                int numEdgesPerThread,
                                uint *dist,
                                uint *preNode,
                                uint *edgesSource,
                                uint *edgesEnd,
                                uint *edgesWeight,
                                bool *finished) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startId = threadId * numEdgesPerThread;

    if (startId >= numEdges) {
        return;
    }

    int endId = (threadId + 1) * numEdgesPerThread;
    if (endId >= numEdges) {
        endId = numEdges;
    }

    for (int nodeId = startId; nodeId < endId; nodeId++) {
        uint source = edgesSource[nodeId];
        uint end = edgesEnd[nodeId];
        uint weight = edgesWeight[nodeId];

        if (dist[source] + weight < dist[end]) {
        atomicMin(&dist[end], dist[source] + weight);
        // dist[end] = dist[source] + weight;
        preNode[end] = source;
        *finished = false;
        }
    }

}

uint* sssp_GPU(Graph *graph, int source) {
    int numNodes = graph->numNodes;
    int numEdges = graph->numEdges;
    uint *dist = new uint[numNodes];
    uint *preNode = new uint[numNodes];
    uint *edgesSource = new uint[numEdges];
    uint *edgesEnd = new uint[numEdges];
    uint *edgesWeight = new uint[numEdges];

    for (int i = 0; i < numNodes; i++) {
        dist[i] = MAX_DIST;
        preNode[i] = uint(-1);
    }


    for (int i = 0; i < numEdges; i++) {
        Edge edge = graph->edges.at(i);

        // Transfer the vector to the following three arrays
        edgesSource[i] = edge.source;
        edgesEnd[i] = edge.end;
        edgesWeight[i] = edge.weight;

        if (edge.source == source){
            if (edge.weight < dist[edge.end]){
                dist[edge.end] = edge.weight;
                preNode[edge.end] = source;
            }
        } else {
        // Case: edge.source != source
        continue;
        }
    }

    dist[source] = 0;
    preNode[source] = 0;


    uint *d_dist;
    uint *d_preNode;
    bool *d_finished;
    uint *d_edgesSource;
    uint *d_edgesEnd;
    uint *d_edgesWeight;

    gpuErrorcheck(hipMalloc(&d_dist, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_preNode, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_edgesSource, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesEnd, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesWeight, numEdges * sizeof(uint)));

    gpuErrorcheck(hipMemcpy(d_dist, dist, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_preNode, preNode, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesSource, edgesSource, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesEnd, edgesEnd, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesWeight, edgesWeight, numEdges * sizeof(uint), hipMemcpyHostToDevice));

    Timer timer;
    int numIteration = 0;
    int numEdgesPerThread = 8;
    int numThreadsPerBlock = 512;
    int numBlock = (numEdges) / (numThreadsPerBlock * numEdgesPerThread) + 1;
    bool finished = true;

    timer.start();
    do {
        numIteration++;
        finished = true;

        gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

        // TO-DO PARALLEL
        sssp_GPU_Kernel<<< numBlock, numThreadsPerBlock >>> (numEdges,
                                        numEdgesPerThread,
                                        d_dist,
                                        d_preNode,
                                        d_edgesSource,
                                        d_edgesEnd,
                                        d_edgesWeight,
                                        d_finished);

        gpuErrorcheck(hipPeekAtLastError());
        gpuErrorcheck(hipDeviceSynchronize()); 
        gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
    } while(!finished);
    timer.stop();


    printf("Process Done!\n");
    printf("Number of Iteration: %d\n", numIteration);
    printf("The execution time of SSSP on GPU: %f ms\n", timer.elapsedTime());

    gpuErrorcheck(hipMemcpy(dist, d_dist, numNodes * sizeof(uint), hipMemcpyDeviceToHost));

    gpuErrorcheck(hipFree(d_dist));
    gpuErrorcheck(hipFree(d_preNode));
    gpuErrorcheck(hipFree(d_finished));
    gpuErrorcheck(hipFree(d_edgesSource));
    gpuErrorcheck(hipFree(d_edgesEnd));
    gpuErrorcheck(hipFree(d_edgesWeight));

    return dist;
}

__global__ void sssp_GPU_Hybrid_Message(int numNodesPerThread,
                                    uint *dist,
                                    uint *preNode,
                                    int *d_messageToDeviceSize,
                                    int *d_messageToDeviceNodeIndex,
                                    int *d_messageToDeviceDist) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startId = threadId * numNodesPerThread;
    if (startId >= *d_messageToDeviceSize) {
        return;
    }

    int endId = (threadId + 1) * numNodesPerThread;
    if (endId >= *d_messageToDeviceSize) {
        endId = *d_messageToDeviceSize;
    }

    for (int i = startId; i < endId; i++) {
        int nodeId = d_messageToDeviceNodeIndex[i];
        int updateDist = d_messageToDeviceDist[i];
        if (dist[nodeId] > updateDist) {
            atomicMin(&dist[nodeId], updateDist);
        }
    }
}



__global__ void sssp_GPU_Hybrid_Kernel(int splitIndex,
                                int numEdges,
                                int numEdgesPerThread,
                                uint *dist,
                                uint *preNode,
                                uint *edgesSource,
                                uint *edgesEnd,
                                uint *edgesWeight,
                                bool *finished,
                                int *d_messageToHostSize,
                                int *d_messageToHostNodeIndex,
                                int *d_messageToHostDist) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startId = splitIndex + threadId * numEdgesPerThread;
    if (startId >= numEdges) {
        return;
    }

    int endId = splitIndex + (threadId + 1) * numEdgesPerThread;
    if (endId >= numEdges) {
        endId = numEdges;
    }

    // printf("GPU: process edged from: %d to %d \n", startId, endId);
    for (int nodeId = startId; nodeId < endId; nodeId++) {
        uint source = edgesSource[nodeId];
        uint end = edgesEnd[nodeId];
        uint weight = edgesWeight[nodeId];
        
        uint tempDist = dist[source] + weight;
        if (tempDist < dist[end]) {
            atomicMin(&dist[end], tempDist);
            preNode[end] = source;
            *finished = false;

            // Message
            // int index = *d_messageToHostSize++;
            int index = atomicAdd(d_messageToHostSize, 1);
            d_messageToHostNodeIndex[index] = end;
            d_messageToHostDist[index] = tempDist;
            // printf("GPUUUUUUUU Message index: %d\n", index);
        }
    }
}

uint* sssp_Hybrid(Graph *graph, int source) {
    int numNodes = graph->numNodes;
    int numEdges = graph->numEdges;
    uint *dist = new uint[numNodes];
    uint *preNode = new uint[numNodes];
    uint *edgesSource = new uint[numEdges];
    uint *edgesEnd = new uint[numEdges];
    uint *edgesWeight = new uint[numEdges];

    // Message from host to device
    int *messageToDeviceNodeIndex = new int[numEdges];
    int *messageToDeviceDist = new int[numEdges];
    int messageToDeviceSize = 0;
    // Message from device to host
    int *messageToHostNodeIndex  = new int[numEdges];
    int *messageToHostDist = new int[numEdges];
    int messageToHostSize = 0;

    for (int i = 0; i < numNodes; i++) {
        dist[i] = MAX_DIST;
        preNode[i] = uint(-1);

        messageToDeviceNodeIndex[i] = -1;
        messageToDeviceDist[i] = -1;
        messageToHostNodeIndex[i] = -1;
        messageToHostDist[i] = -1;
    }


    for (int i = 0; i < numEdges; i++) {
        Edge edge = graph->edges.at(i);
        
        // Transfer the vector to the following three arrays
        edgesSource[i] = edge.source;
        edgesEnd[i] = edge.end;
        edgesWeight[i] = edge.weight;

        if (edge.source == source){
            if (edge.weight < dist[edge.end]){
                dist[edge.end] = edge.weight;
                preNode[edge.end] = source;
            }
        } else {
            // Case: edge.source != source
            continue;
        }
    }

    dist[source] = 0;
    preNode[source] = 0;

    uint *d_dist;
    uint *d_preNode;
    bool *d_finished;
    uint *d_edgesSource;
    uint *d_edgesEnd;
    uint *d_edgesWeight;
    int *d_messageToHostNodeIndex;
    int *d_messageToHostDist;
    int *d_messageToHostSize;
    int *d_messageToDeviceNodeIndex;
    int *d_messageToDeviceDist;
    int *d_messageToDeviceSize;

    gpuErrorcheck(hipMalloc(&d_dist, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_preNode, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_edgesSource, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesEnd, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesWeight, numEdges * sizeof(uint)));
    
    gpuErrorcheck(hipMalloc(&d_messageToHostNodeIndex, numEdges * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_messageToHostDist, numEdges * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_messageToHostSize, sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_messageToDeviceNodeIndex, numEdges * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_messageToDeviceDist, numEdges * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_messageToDeviceSize, sizeof(int)));


    gpuErrorcheck(hipMemcpy(d_dist, dist, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_preNode, preNode, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesSource, edgesSource, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesEnd, edgesEnd, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesWeight, edgesWeight, numEdges * sizeof(uint), hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemcpy(d_messageToHostNodeIndex, messageToHostNodeIndex, numEdges * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_messageToHostDist, messageToHostDist, numEdges * sizeof(int), hipMemcpyHostToDevice));



    Timer timer;
    int numIteration = 0;
    bool finished = false;
    bool h_finished = false;
    
    
    float splitRatio; // cpu_data_size / whole_data_size

    // Automatic select a prior value of spritRatio based on experience
    if (numEdges < 300000) {
        splitRatio = 0.95;
    } else if (numEdges < 800000) {
        splitRatio = 0.7;
    } else {
        splitRatio = 0.1;
    }

    /*
    CPU process edges from 0 to splitIndex   
        number of edges: splitIndex
    GPU process edges from splitIndex to numEdges 
        number of edges: numEdges - splitIndex + 1
    */
    int splitIndex = numEdges * splitRatio;
    int d_numEdgesPerThread = 8;
    // int d_numNodesPerThread = 8;
    int d_numThreadsPerBlock = 512;
    int d_numBlock = (numEdges - splitIndex + 1) / (d_numThreadsPerBlock * d_numEdgesPerThread) + 1;
    
    Timer timer_cpu, timer_gpu;
    Timer timer_host_to_device;
    Timer timer_device_to_host;

    // Default: enable cpu and gpu 
    // Once splitRatio equals to 0 only enable gpu
    // Once splitRatio equals to 1 only enable cpu
    
    bool cpu_enable = true;
    bool gpu_enable = true;

    vector<LoopInfo> infos;
    LoopInfo loopInfo;

    





    timer.start();
    do {
        // splitRatio = 0.4;

        numIteration++;
        finished = true;
        h_finished = true;
        splitIndex = numEdges * splitRatio;
        d_numBlock = (numEdges - splitIndex + 1) / (d_numThreadsPerBlock * d_numEdgesPerThread) + 1;
        
        messageToDeviceSize = 0;
        messageToHostSize = 0;


        timer_gpu.start();
        timer_cpu.start();
        #pragma omp parallel //num_threads(8)
        {   
            int threadId = omp_get_thread_num();
            int h_numThreads = omp_get_num_threads();
            if (threadId == h_numThreads - 1 && splitIndex < numEdges  && gpu_enable) {
                // Last thread will be used to launch gpu kernel 
                // if thread 0 is used to launch gpu kernel, the first block of 
                // data whose index begining from 0 will not be processed.
                gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
                // timer_host_to_device.start();
                // gpuErrorcheck(hipMemcpy(d_dist, dist, sizeof(uint) * numNodes, hipMemcpyHostToDevice));
                // timer_host_to_device.stop();
                gpuErrorcheck(hipMemcpy(d_messageToHostSize, &messageToHostSize, sizeof(int), hipMemcpyHostToDevice));

                sssp_GPU_Hybrid_Kernel<<< d_numBlock, d_numThreadsPerBlock>>> (splitIndex,
                                                                        numEdges,
                                                                        d_numEdgesPerThread,
                                                                        d_dist,
                                                                        d_preNode,
                                                                        d_edgesSource,
                                                                        d_edgesEnd,
                                                                        d_edgesWeight,
                                                                        d_finished,
                                                                        d_messageToHostSize,
                                                                        d_messageToHostNodeIndex,
                                                                        d_messageToHostDist);
                gpuErrorcheck(hipPeekAtLastError());
                gpuErrorcheck(hipDeviceSynchronize()); 
                gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
                // timer_device_to_host.start();
                // gpuErrorcheck(hipMemcpy(dist_copy, d_dist, sizeof(uint) * numNodes, hipMemcpyDeviceToHost));
                // timer_device_to_host.stop();

                gpuErrorcheck(hipMemcpy(&messageToHostSize, d_messageToHostSize, sizeof(int), hipMemcpyDeviceToHost));
                int messageSize = messageToHostSize;
                gpuErrorcheck(hipMemcpy(messageToHostNodeIndex, d_messageToHostNodeIndex, sizeof(int) * messageSize, hipMemcpyDeviceToHost));
                gpuErrorcheck(hipMemcpy(messageToHostDist, d_messageToHostDist, sizeof(int) * messageSize, hipMemcpyDeviceToHost));

                /* printf("messageToHost size: %d\n", messageSize);
                for (int i = 0; i < messageSize; i++) {
                    printf("index: %d dist: %d\n", messageToHostNodeIndex[i], messageToHostDist[i]);
                } */

                timer_gpu.stop();
            } else if (cpu_enable && threadId != (h_numThreads - 1)) {
                // printf("Sub threads\n");
                int h_numEdgesPerThread = (splitIndex) / (h_numThreads - 1) + 1;
                int start = threadId * h_numEdgesPerThread;
                int end = (threadId + 1) * h_numEdgesPerThread;
                if (start > splitIndex) {
                    start = splitIndex;
                }
                if (end > splitIndex) {
                    end = splitIndex;
                }

                // cout << "Processs node: from " << start << " to: " << end << endl;
                // printf("Process node from: %d to : %d\n", start, end);
                for (int i = start; i < end; i++) {
                    uint source = edgesSource[i];
                    uint end = edgesEnd[i];
                    uint weight = edgesWeight[i];
                    
                    uint tempDist = dist[source] + weight;
                    if (tempDist < dist[end]) {
                        dist[end] = tempDist;
                        preNode[end] = source;
                    // New message  
                    #pragma omp critical
                    {
                        messageToDeviceNodeIndex[messageToDeviceSize] = end;
                        messageToDeviceDist[messageToDeviceSize] = tempDist;
                        messageToDeviceSize++;
                    }            
                    h_finished = false;
                    }
                }
                timer_cpu.stop();
            }
        }
       
        finished = finished && h_finished;

        #pragma omp parallel //num_threads(8)
        {   
            int threadId = omp_get_thread_num();
            int h_numThreads = omp_get_num_threads();
            if (threadId == h_numThreads - 1) {
                // Process message from host in GPU
                int d_numNodesPerThread = 8;
                /* if (messageToDeviceSize > numNodes) {
                    messageToDeviceSize = numNodes;
                } */
                d_numBlock = (messageToDeviceSize) / (d_numThreadsPerBlock * d_numNodesPerThread) + 1;
                gpuErrorcheck(hipMemcpy(d_messageToDeviceSize, &messageToDeviceSize, sizeof(int), hipMemcpyHostToDevice));
                gpuErrorcheck(hipMemcpy(d_messageToDeviceNodeIndex, messageToDeviceNodeIndex, sizeof(int) * messageToDeviceSize, hipMemcpyHostToDevice));
                gpuErrorcheck(hipMemcpy(d_messageToDeviceDist, messageToDeviceDist, sizeof(int) * messageToDeviceSize, hipMemcpyHostToDevice));

                sssp_GPU_Hybrid_Message<<< d_numBlock, d_numThreadsPerBlock>>> (d_numNodesPerThread,
                                                                            d_dist,
                                                                            d_preNode,
                                                                            d_messageToDeviceSize,
                                                                            d_messageToDeviceNodeIndex,
                                                                            d_messageToDeviceDist);

                gpuErrorcheck(hipPeekAtLastError());
                gpuErrorcheck(hipDeviceSynchronize());    
            } else if (threadId != h_numThreads - 1) {
                int h_numNodesPerThread = (messageToHostSize) / (h_numThreads - 1) + 1;
                int start = threadId * h_numNodesPerThread;
                int end = (threadId + 1) * h_numNodesPerThread;
                if (start > messageToHostSize) {
                    start = messageToHostSize;
                }
                if (end > messageToHostSize) {
                    end = messageToHostSize;
                }
                for (int i = start; i < end; i++) {
                    int nodeId = messageToHostNodeIndex[i];
                    int updateDist = messageToHostDist[i];
                    if (dist[nodeId] > updateDist) {
                        dist[nodeId] = updateDist;
                    }
                }
            }
        }

        // Load Balancing

        if (cpu_enable && gpu_enable) {
            float factor = (timer_cpu.elapsedTime() / timer_gpu.elapsedTime());
            if (factor > 1.1) {
                splitRatio = splitRatio - 0.05;
                if (splitRatio < 0) {
                    splitRatio = 0;
                    cpu_enable = false;
                }
    
            } else if (factor < 0.9) {
                splitRatio = splitRatio + 0.05;
                if (splitRatio > 1) {
                    splitRatio = 1;
                    gpu_enable = false;
                }
            }

            // printf("No. itr: %d , updated splitRatio: %f, factor: %f\n", numIteration, splitRatio, factor);

            // printf("CPU PART TIME: %f\n", timer_cpu.elapsedTime());
            // printf("GPU PART TIME: %f\n", timer_gpu.elapsedTime());
            // printf("Copy dist from host to device : %f ms \n", timer_host_to_device.elapsedTime());
            // printf("Copy dist from device to host : %f ms \n", timer_device_to_host.elapsedTime());
            loopInfo.numIteration = numIteration;
            loopInfo.time_cpu = timer_cpu.elapsedTime() > 0 ? timer_cpu.elapsedTime() : 0;
            loopInfo.time_gpu = timer_gpu.elapsedTime() > 0 ? timer_gpu.elapsedTime() : 0;
            loopInfo.splitRatio = splitRatio;
            infos.push_back(loopInfo);
        } 
    } while(!finished);
    timer.stop();

    printLoopInfo(infos);
    printf("Process Done!\n");
    printf("Number of Iteration: %d\n", numIteration);
    printf("The execution time of SSSP on Hybrid(CPU-GPU): %f ms\n", timer.elapsedTime());

    gpuErrorcheck(hipFree(d_dist));
    gpuErrorcheck(hipFree(d_preNode));
    gpuErrorcheck(hipFree(d_finished));
    gpuErrorcheck(hipFree(d_edgesSource));
    gpuErrorcheck(hipFree(d_edgesEnd));
    gpuErrorcheck(hipFree(d_edgesWeight));
    gpuErrorcheck(hipFree(d_messageToHostNodeIndex));
    gpuErrorcheck(hipFree(d_messageToHostDist));
    gpuErrorcheck(hipFree(d_messageToHostSize));
    gpuErrorcheck(hipFree(d_messageToDeviceNodeIndex));
    gpuErrorcheck(hipFree(d_messageToDeviceDist));
    gpuErrorcheck(hipFree(d_messageToDeviceSize));

    return dist;
}



int main(int argc, char **argv) {
    Timer timer_total, timer_load;
    timer_total.start();
    
    ArgumentParser args(argc, argv);

    timer_load.start();
    Graph graph(args.inputFilePath);
    //Graph graph("datasets/simpleGraph.txt");
    graph.readGraph();
    timer_load.stop();
    

    int sourceNode;

    if (args.hasSourceNode) {
        sourceNode = args.sourceNode;
    } else {
        // Use graph default source 
        sourceNode = graph.defaultSource;
    }


    uint *dist_hybrid = sssp_Hybrid(&graph, sourceNode);
    uint *dist_gpu = sssp_GPU(&graph, sourceNode);

    compareResult(dist_hybrid, dist_gpu, graph.numNodes);

    if (args.runOnCPU) {
        uint *dist_cpu = sssp_CPU_parallel(&graph, sourceNode);
        compareResult(dist_cpu, dist_hybrid, graph.numNodes);
    }

    timer_total.stop();
    printf("Total execution time: %f ms\n", timer_total.elapsedTime());
    printf("Graph loading execution time: %f ms\n", timer_load.elapsedTime());

    return 0;
}