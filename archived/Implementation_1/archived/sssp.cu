#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "./utilities/timer.hpp"
#include "./utilities/graph.hpp"
#include "./utilities/gpu_error_check.cuh"

/*

Version 1. 1 Node Per Thread



*/

// const int MAX_DIST = 65535; //  initial value of distance
int numNodes;
int numEdges;

int* dist;
int* previousNode;
// int dist[NUMBER];   //  array to store the distance from source to each nodes
// int previousNode[NUMBER];   //  
// int graph[NUMBER][NUMBER];  //  a matrix to represent the graph
int ** graph;
bool* finished;
int* graph_static;

void init(Graph* graphData, int source) {
    numNodes = graphData->numNodes;
    graph = graphData->graph;

    int size = numNodes;

    dist = new int[size];
    previousNode = new int[size];
    finished = new bool[size];

    for (int i = 0; i < size; i++) {
        // set the distance to the source node
        dist[i] = graph[source][i];
        finished[i] = false;
        if (dist[i] == MAX_DIST) {
            previousNode[i] = -1;
        }
        else {
            previousNode[i] = source;
        }
    }
    // set dist[source] = 0
    dist[source] = 0;

}

void printShortestDistance(int source) {
    int diameter = 0;
    for (int i = 0; i < numNodes; i++) {
        if(dist[i] != MAX_DIST){
            if(dist[i] > diameter){
                diameter = dist[i];
            }
            // printf("Shortest distance from node: %d to source: %d: is: %d\n", i, source, dist[i]);
        }else{
            // printf("Shortest distance from node: %d to source: %d: is: INF\n", i, source);
        } 
    }
    printf("Diameter: %d\n", diameter);
}

void printGraph(){
    printf("\n\nGraph");
    for (int i = 0; i < numNodes; i++){
        for (int j = 0; j < numNodes; j++){
            printf("%d ", graph[i][j]);
        }
        printf("\n");
    }
}

void printFinished(){
    printf("Finished array\n");
    for(int i = 0; i < numNodes; i++){
        printf("Node: %d, status: %d\n", i, finished[i]);
    }
    printf("\n");
}

void dijkstraOnCPU(int source) {
    Timer timer;
    int size = numNodes;
    
    // Find the connected nodes to the source point
    // Set the source point
    dist[source] = 0;
    finished[source] = true;

    timer.start();
    for (int i = 0; i < size; i++) {
        int mindist = MAX_DIST;
        // U is the closet point to source, u is not finished yet
        int u = source;

        for (int j = 0; j < size; j++) {
            if ((j != u) && (!finished[j]) && dist[j] < mindist) {
                u = j;
                mindist = dist[j];
            }
        }

        finished[u] = true;

        for (int j = 0; j < size; j++) {
            if ((j != u) && (!finished[j]) && graph[u][j] < MAX_DIST) {
                // Find the shorter path
                if (dist[u] + graph[u][j] < dist[j]) {
                    // Update dist
                    dist[j] = dist[u] + graph[u][j];
                    // Update its previous point
                    previousNode[j] = u;
                }
            }
        }
    }
    printf("The execution time of SSSP on CPU: %d ms\n", timer.stop());
    // printShortestDistance(0);
}

__global__ void dijkstraOnGPU_kernel1(int numNodes, 
                                int sourceId,
                                int* graphData,
                                bool* finished,
                                int* dist,
                                int* prev,
                                int* closestNodeId,
                                int* minimumDist,
                                bool* completed) {
    // kernel to compute the closest node 
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int nodeId = threadId;
    //int nodeBeginId = partSize * partId;
    //int nodeEndId = partSize * (partId + 1);
    //if (nodeEndId > numNodes) nodeEndId = numNodes;

    //printf("Thread %d: processes the nodes from %d to %d\n", partId, nodeBeginId, nodeEndId);
    // printf("Thread: %d", threadId);
    if (nodeId < numNodes){
        // printf("This thread id is: %d\n", threadId);
        // printf("dist[%d] is %d, and the closest distance is %d\n\n", nodeId, dist[nodeId], dist[*closestNodeId]);
        if (!finished[nodeId] && dist[nodeId] < *minimumDist){
            // printf("Finished?");
            *closestNodeId = nodeId;
            *minimumDist = dist[nodeId];
            // printf("updated closetNodeId: %d\n", *closestNodeId);
            *completed = false;
        }
    }
    // printf("Graph[0][2]: %d\n", graphData[2]);
    /* printf("Print graph from GPU. Num nodes: %d \n", numNodes);
    for(int i = 0; i < numNodes; i++){
        for(int j = 0; j < numNodes; j++){
            printf("[%d][%d]: %d ", i, j, graphData[i][j]);
        }
        printf("\n");
    }
    printf("\n"); */


}

__global__ void dijkstraOnGPU_kernel2(int numNodes, 
                                        int sourceId,
                                        int* graphData,
                                        bool* finished,
                                        int* dist,
                                        int* prev,
                                        int* closestNodeId,
                                        int GRAPH_MAX_SIZE) {

    // Based on closest node then update its connected node
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int nodeId = threadId;
    if(nodeId > numNodes) return;
    // rowIndex = closestNodeId;
    // colIndex = nodeId;
    // Convert to 1-D index
    int index = *closestNodeId * GRAPH_MAX_SIZE + nodeId;
    finished[*closestNodeId] = true;
    // printf("graphData[%d][%d]: is %d \n", *closestNodeId, nodeId, graphData[*closestNodeId][nodeId]);
    if (!finished[nodeId] && graphData[index] < MAX_DIST){
        // Find the shorter path
        if(dist[*closestNodeId] + graphData[index] < dist[nodeId]){
            // Update dist
            dist[nodeId] = dist[*closestNodeId] + graphData[index];
            // Update its previous point
            prev[nodeId] = *closestNodeId;
            // printf("update prev[%d] = %d\n", nodeId, *closestNodeId);
            // printf("update dist[%d] = %d\n", nodeId, dist[*closestNodeId] + graphData[index]);
        }
    }
}

void dijkstraOnGPU(int source){
    Timer timer;
    hipFree(0);
    // Define CPU vars
    // int* closestNodeId = new int(6);
    int closestNodeId = 6;
    // Define GPU vars
    int* d_graph;   // 2D array is converted to 1-D, row = i / cols, col = i % cols;
    int* d_dist;
    int* d_prev;
    bool* d_finished;
    int* d_closestNodeId;
    int* d_minimumDist;
    bool* d_completed;


    /* int width = GRAPH_MAX_SIZE, height = GRAPH_MAX_SIZE;
    size_t pitch;
    size_t size = sizeof(int) * width;    */

    // gpuErrorcheck(hipMallocPitch((void **)&d_graph, &pitch, size, height));
    // gpuErrorcheck(hipMemset2D(d_graph, pitch, 0, size, height));
    gpuErrorcheck(hipMalloc((void **)&d_graph, GRAPH_MAX_SIZE * GRAPH_MAX_SIZE * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_dist, numNodes * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_prev, numNodes * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_finished, numNodes * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_closestNodeId, sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_completed, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_minimumDist, sizeof(int)));

    // gpuErrorcheck(hipMemcpy2D(d_graph, pitch, graph1.graph, size, size, height, hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_graph, graph[0], GRAPH_MAX_SIZE * GRAPH_MAX_SIZE * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_dist, dist, numNodes * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_prev, previousNode, numNodes * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_finished, finished, numNodes * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_closestNodeId, &closestNodeId, sizeof(int), hipMemcpyHostToDevice));

    bool completed = true;
    int minimumDist = MAX_DIST;
    int numIteration = 0;


    timer.start();

    do{
        numIteration++;
        completed = true;
        minimumDist = MAX_DIST;
        // Each block has 128 threads
        int numThreadPerBlock = 128;
        int numBlock = (numNodes / numThreadPerBlock) + 1;
        
        gpuErrorcheck(hipMemcpy(d_completed, &completed, sizeof(bool), hipMemcpyHostToDevice));
        gpuErrorcheck(hipMemcpy(d_minimumDist, &minimumDist, sizeof(int), hipMemcpyHostToDevice));

        dijkstraOnGPU_kernel1<<<numBlock, numThreadPerBlock >>>(numNodes,
                                                        source,
                                                        d_graph,
                                                        d_finished,
                                                        d_dist,
                                                        d_prev,
                                                        d_closestNodeId,
                                                        d_minimumDist,
                                                        d_completed);

        gpuErrorcheck(hipPeekAtLastError());
        gpuErrorcheck(hipDeviceSynchronize());
        
        dijkstraOnGPU_kernel2<<<numBlock, numThreadPerBlock>>>(numNodes,
                                                        source,
                                                        d_graph,
                                                        d_finished,
                                                        d_dist,
                                                        d_prev,
                                                        d_closestNodeId,
                                                        GRAPH_MAX_SIZE);


        gpuErrorcheck(hipDeviceSynchronize());  
        gpuErrorcheck(hipMemcpy(&completed, d_completed, sizeof(bool), hipMemcpyDeviceToHost));
        gpuErrorcheck(hipMemcpy(finished, d_finished, numNodes * sizeof(bool), hipMemcpyDeviceToHost));
        // printFinished();
        // printf("finished: %d\n", completed);
    }while(!completed);

    printf("Number of Iteration Executed: %d\n", numIteration);
    printf("The execution time of SSSP on GPU: %d ms\n", timer.stop());
    // print("%d", d_closestNodeId);
    hipMemcpy(&closestNodeId, d_closestNodeId, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(dist, d_dist, numNodes * sizeof(int), hipMemcpyDeviceToHost);
    // printf("%d", (int)(*closestNodeId));
    // printf("%d", closestNodeId);

    // printGraph();

    
    // graph.printGraph();

    hipFree(d_graph);
    hipFree(d_dist);
    hipFree(d_prev);
    hipFree(d_finished);
    hipFree(d_closestNodeId);
    hipFree(d_minimumDist);
    hipFree(d_completed);

    // printShortestDistance(0);
}

//int main() {
//
//    // Graph graph1("simpleGragh.txt");
//    // Graph graph1("email-Eu-core-SIMPLE.txt");
//    Graph graph1("p2p-Gnutella08.txt");
//    // Graph graph1("email-Eu-core.txt");
//     //Graph graph("testGraph.txt");
//    graph1.readGraph();
//    int sourceId = 0;
//
//    init(&graph1, sourceId);   // source 0
//        
//    // Run SSSP on CPU
//    dijkstraOnCPU(sourceId);
//
//    printShortestDistance(sourceId);
//
//
//    init(&graph1, sourceId);   // source 0
//
//    // Run SSSP on GPU
//    dijkstraOnGPU(sourceId);
//    printShortestDistance(sourceId);
//
//   
//
//    return 0;
//}