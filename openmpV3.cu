#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "./utilities/timer.hpp"
#include "./utilities/graph.hpp"
#include "./utilities/gpu_error_check.cuh"
#include "./utilities/global.hpp"
#include "./utilities/argument_parser.hpp"
#include <omp.h>


uint* sssp_CPU_parallel(Graph *graph, int source) {
    int numNodes = graph->numNodes;
    int numEdges = graph->numEdges;
    uint *dist = new uint[numNodes];
    uint *preNode = new uint[numNodes];
    bool *processed = new bool[numNodes];
    uint *edgesSource = new uint[numEdges];
    uint *edgesEnd = new uint[numEdges];
    uint *edgesWeight = new uint[numEdges];

    for (int i = 0; i < numNodes; i++) {
        dist[i] = MAX_DIST;
        preNode[i] = uint(-1);
        processed[i] = false;
    }

    for (int i = 0; i < numEdges;i ++) {
        Edge edge = graph->edges.at(i);
        edgesSource[i] = edge.source;
        edgesEnd[i] = edge.end;
        edgesWeight[i] = edge.weight;

        if (edge.source == source) {
            if (edge.weight < dist[edge.end]) {
                dist[edge.end] = edge.weight;
                preNode[edge.end] = source;
            }
        } else {
            // Case: edge.source != source
            continue;
        }
    }
    
    Timer timer;
    bool finished = false;
    uint numIteration = 0;

    dist[source] = 0;
    preNode[source] = 0;
    processed[source] = true;

    timer.start();
    while(!finished) {
        finished = true;
        numIteration++;
        
        #pragma omp parallel 
        {   
            // #pragma omp master 
            int threadId = omp_get_thread_num();
            int numThreads = omp_get_num_threads();
            int numEdgesPerThread = numEdges / numThreads + 1;
            int start = threadId * numEdgesPerThread;
            int end = (threadId + 1) * numEdgesPerThread;
            if (start > numEdges) {
                start = numEdges;
            }
            
            if (end > numEdges) {
                end = numEdges;
            }

            for (int i = start; i < end; i++) {
                uint source = edgesSource[i];
                uint end = edgesEnd[i];
                uint weight = edgesWeight[i];

                if (dist[source] + weight < dist[end]) {
                    // #pragma omp atomic
                    dist[end] = dist[source] + weight;
                    // #pragma omp atomic
                    preNode[end] = source;
                    finished = false;
                }
            }
        }
    }
    timer.stop();

    printf("Process Done!\n");
    printf("Number of Iteration: %d\n", numIteration);
    printf("The execution time of SSSP on CPU(OpenMP): %f ms\n", timer.elapsedTime());

    return dist;

}

__global__ void sssp_GPU_Kernel(int numEdges,
                                int numEdgesPerThread,
                                uint *dist,
                                uint *preNode,
                                uint *edgesSource,
                                uint *edgesEnd,
                                uint *edgesWeight,
                                bool *finished) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startId = threadId * numEdgesPerThread;

    if (startId >= numEdges) {
        return;
    }

    int endId = (threadId + 1) * numEdgesPerThread;
    if (endId >= numEdges) {
        endId = numEdges;
    }

    for (int nodeId = startId; nodeId < endId; nodeId++) {
        uint source = edgesSource[nodeId];
        uint end = edgesEnd[nodeId];
        uint weight = edgesWeight[nodeId];

        if (dist[source] + weight < dist[end]) {
        atomicMin(&dist[end], dist[source] + weight);
        // dist[end] = dist[source] + weight;
        preNode[end] = source;
        *finished = false;
        }
    }

}

uint* sssp_GPU(Graph *graph, int source) {
    int numNodes = graph->numNodes;
    int numEdges = graph->numEdges;
    uint *dist = new uint[numNodes];
    uint *preNode = new uint[numNodes];
    uint *edgesSource = new uint[numEdges];
    uint *edgesEnd = new uint[numEdges];
    uint *edgesWeight = new uint[numEdges];

    for (int i = 0; i < numNodes; i++) {
        dist[i] = MAX_DIST;
        preNode[i] = uint(-1);
    }


    for (int i = 0; i < numEdges; i++) {
        Edge edge = graph->edges.at(i);

        // Transfer the vector to the following three arrays
        edgesSource[i] = edge.source;
        edgesEnd[i] = edge.end;
        edgesWeight[i] = edge.weight;

        if (edge.source == source){
            if (edge.weight < dist[edge.end]){
                dist[edge.end] = edge.weight;
                preNode[edge.end] = source;
            }
        } else {
        // Case: edge.source != source
        continue;
        }
    }

    dist[source] = 0;
    preNode[source] = 0;


    uint *d_dist;
    uint *d_preNode;
    bool *d_finished;
    uint *d_edgesSource;
    uint *d_edgesEnd;
    uint *d_edgesWeight;

    gpuErrorcheck(hipMalloc(&d_dist, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_preNode, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_edgesSource, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesEnd, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesWeight, numEdges * sizeof(uint)));

    gpuErrorcheck(hipMemcpy(d_dist, dist, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_preNode, preNode, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesSource, edgesSource, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesEnd, edgesEnd, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesWeight, edgesWeight, numEdges * sizeof(uint), hipMemcpyHostToDevice));

    Timer timer;
    int numIteration = 0;
    int numEdgesPerThread = 8;
    int numThreadsPerBlock = 512;
    int numBlock = (numEdges) / (numThreadsPerBlock * numEdgesPerThread) + 1;
    bool finished = true;

    timer.start();
    do {
        numIteration++;
        finished = true;

        gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

        // TO-DO PARALLEL
        sssp_GPU_Kernel<<< numBlock, numThreadsPerBlock >>> (numEdges,
                                        numEdgesPerThread,
                                        d_dist,
                                        d_preNode,
                                        d_edgesSource,
                                        d_edgesEnd,
                                        d_edgesWeight,
                                        d_finished);

        gpuErrorcheck(hipPeekAtLastError());
        gpuErrorcheck(hipDeviceSynchronize()); 
        gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
    } while(!finished);
    timer.stop();


    printf("Process Done!\n");
    printf("Number of Iteration: %d\n", numIteration);
    printf("The execution time of SSSP on GPU: %f ms\n", timer.elapsedTime());

    gpuErrorcheck(hipMemcpy(dist, d_dist, numNodes * sizeof(uint), hipMemcpyDeviceToHost));

    gpuErrorcheck(hipFree(d_dist));
    gpuErrorcheck(hipFree(d_preNode));
    gpuErrorcheck(hipFree(d_finished));
    gpuErrorcheck(hipFree(d_edgesSource));
    gpuErrorcheck(hipFree(d_edgesEnd));
    gpuErrorcheck(hipFree(d_edgesWeight));

    return dist;
}


__global__ void sssp_Hybrid_GPU_Kernel(int splitIndex,
                                int numEdges,
                                int numEdgesPerThread,
                                uint *dist,
                                uint *preNode,
                                uint *edgesSource,
                                uint *edgesEnd,
                                uint *edgesWeight,
                                bool *finished,
                                uint *d_msgToHostIndex,
                                uint *d_msgToHostNodeId,
                                uint *d_msgToHostDist) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startId = splitIndex + threadId * numEdgesPerThread;
    if (startId >= numEdges) {
        return;
    }

    int endId = splitIndex + (threadId + 1) * numEdgesPerThread;
    if (endId >= numEdges) {
        endId = numEdges;
    }

    // printf("GPU: process edged from: %d to %d \n", startId, endId);
    for (int nodeId = startId; nodeId < endId; nodeId++) {
        uint source = edgesSource[nodeId];
        uint end = edgesEnd[nodeId];
        uint weight = edgesWeight[nodeId];
        
        if (dist[source] + weight < dist[end]) {
            atomicMin(&dist[end], dist[source] + weight);
            preNode[end] = source;
            *finished = false;

            int index = atomicAdd(d_msgToHostIndex, 1);
            d_msgToHostNodeId[index] = end;
            d_msgToHostDist[index] = dist[end];
        }
    }
}

void sssp_Hybrid_CPU(int threadId,
                    int splitIndex,
                    int numEdges,
                    int numEdgesPerThread,
                    uint *dist,
                    uint *preNode,
                    uint *edgesSource,
                    uint *edgesEnd,
                    uint *edgesWeight,
                    bool *finished,
                    uint *msgToDeviceIndex,
                    uint *msgToDeviceNodeId,
                    uint *msgToDeviceDist) {
    int start = threadId * numEdgesPerThread;
    int end = (threadId + 1) * numEdgesPerThread;
    if (start > splitIndex) return;
    if (end > splitIndex) {
        end = splitIndex;
    }

    for (int i = start; i < end; i++) {
        uint source = edgesSource[i];
        uint end = edgesEnd[i];
        uint weight = edgesWeight[i];
        
        if (dist[source] + weight < dist[end]) { 
            
            dist[end] = dist[source] + weight;
            preNode[end] = source;

            *finished = false;

            uint index;
            
            #pragma omp critical
            {
                index = *msgToDeviceIndex;
                *msgToDeviceIndex = *msgToDeviceIndex + 1;
            
               
                    /* #pragma omp atomic capture
                    {
                        index = *msgToDeviceIndex;
                        *msgToDeviceIndex+=1;
                    } */
                    
                    
                    
                    // printf("index:%d nodeId: %d dist: %d\n", index, end, dist[end]);
            }
            msgToDeviceNodeId[index] = end;
            msgToDeviceDist[index] = dist[end];
        }
    }
}

void sssp_Hybrid_MergeDist(int threadId,
                        int numNodes,
                        int numNodesPerThread,
                        uint *dist,
                        uint *dist_copy) {
    int start = threadId * numNodesPerThread;
    int end = (threadId + 1) * numNodesPerThread;
    if (start > numNodes) return;
    if (end > numNodes) {
        end = numNodes;
    }
    for (int i = start; i < end; i++) {
        if (dist[i] > dist_copy[i]) {
            dist[i] = dist_copy[i];
        }
    }
}

void sssp_Hybrid_Host_Process_Message(int threadId,
                                    int numMsg,
                                    int numMsgPerThread,
                                    uint *dist,
                                    uint *msgToHostNodeId,
                                    uint *msgToHostDist) {
    int start = threadId * numMsgPerThread;
    int end = (threadId + 1) * numMsgPerThread;
    if (start > numMsg) return;
    if (end > numMsg) {
        end = numMsg;
    }
    for (int i = start; i < end; i++) {
        int nodeId = msgToHostNodeId[i];
        int updateDist = msgToHostDist[i];
        #pragma omp critical    
        {
            if (dist[nodeId] > updateDist) {
                dist[nodeId] = updateDist;
            }
        }
    }
}

__global__ void sssp_Hybrid_GPU_Process_Message(int numMsgPerThread,
                                            uint numMsg,
                                            uint *dist,
                                            uint *preNode,
                                            uint *d_msgToDeviceNodeId,
                                            uint *d_msgToDeviceDist) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int start = threadId * numMsgPerThread;
    int end = (threadId + 1) * numMsgPerThread;
    if (start > numMsg) return;
    if (end > numMsg) {
        end = numMsg;
    }
    for (int i = start; i < end; i++) {
        int nodeId = d_msgToDeviceNodeId[i];
        int updateDist = d_msgToDeviceDist[i];
        if (dist[nodeId] > updateDist) {
            atomicMin(&dist[nodeId], updateDist);
        }  
    }

}

uint* sssp_Hybrid(Graph *graph, int source) {
    int numNodes = graph->numNodes;
    int numEdges = graph->numEdges;
    uint *dist = new uint[numNodes];
    uint *preNode = new uint[numNodes];
    uint *edgesSource = new uint[numEdges];
    uint *edgesEnd = new uint[numEdges];
    uint *edgesWeight = new uint[numEdges];
    uint *dist_copy = new uint[numNodes];

    uint *msgToHostNodeId = new uint[numEdges];
    uint *msgToHostDist = new uint[numEdges];
    uint msgToHostIndex = 0;

    uint *msgToDeviceNodeId = new uint[numEdges];
    uint *msgToDeviceDist = new uint[numEdges];
    uint msgToDeviceIndex = 0;

    for (int i = 0; i < numNodes; i++) {
        dist[i] = MAX_DIST;
        preNode[i] = uint(-1);
    }


    for (int i = 0; i < numEdges; i++) {
        Edge edge = graph->edges.at(i);
        
        // Transfer the vector to the following three arrays
        edgesSource[i] = edge.source;
        edgesEnd[i] = edge.end;
        edgesWeight[i] = edge.weight;

        if (edge.source == source){
            if (edge.weight < dist[edge.end]){
                dist[edge.end] = edge.weight;
                preNode[edge.end] = source;
            }
        } else {
            // Case: edge.source != source
            continue;
        }
    }

    dist[source] = 0;
    preNode[source] = 0;

    uint *d_dist;
    uint *d_preNode;
    bool *d_finished;
    uint *d_edgesSource;
    uint *d_edgesEnd;
    uint *d_edgesWeight;
    uint *d_msgToHostNodeId;
    uint *d_msgToHostDist;
    uint *d_msgToHostIndex;
    uint *d_msgToDeviceNodeId;
    uint *d_msgToDeviceDist;
    uint *d_msgToDeviceIndex;

    gpuErrorcheck(hipMalloc(&d_dist, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_preNode, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_edgesSource, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesEnd, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesWeight, numEdges * sizeof(uint)));

    gpuErrorcheck(hipMalloc(&d_msgToHostNodeId, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_msgToHostDist, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_msgToHostIndex, sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_msgToDeviceNodeId, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_msgToDeviceDist, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_msgToDeviceIndex, sizeof(uint)));

    gpuErrorcheck(hipMemcpy(d_dist, dist, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_preNode, preNode, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesSource, edgesSource, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesEnd, edgesEnd, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesWeight, edgesWeight, numEdges * sizeof(uint), hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemcpy(d_msgToHostNodeId, msgToHostNodeId, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_msgToHostDist, msgToHostDist, numEdges * sizeof(uint), hipMemcpyHostToDevice));

    // Copy from gpu memory
    memcpy(dist_copy, dist, numNodes * sizeof(uint));

    Timer timer;
    int numIteration = 0;
    bool finished = false;
    bool h_finished = false;
    
    
    float splitRatio; // cpu_data_size / whole_data_size

    // Automatic select a prior value of spritRatio based on experience
    if (numEdges < 300000) {
        splitRatio = 0.95;
    } else if (numEdges < 800000) {
        splitRatio = 0.7;
    } else {
        splitRatio = 0.5;
    }
    /*
    CPU process edges from 0 to splitIndex   
        number of edges: splitIndex
    GPU process edges from splitIndex to numEdges 
        number of edges: numEdges - splitIndex + 1
    */
    int splitIndex = numEdges * splitRatio;
    int d_numEdgesPerThread = 8;
    int d_numThreadsPerBlock = 512;
    int d_numBlock = (numEdges - splitIndex + 1) / (d_numThreadsPerBlock * d_numEdgesPerThread) + 1;
    
    Timer timer_cpu, timer_gpu;
    Timer timer_cpu_message;
    Timer timer_gpu_message;

    // Default: enable cpu and gpu 
    // Once splitRatio equals to 0 only enable gpu
    // Once splitRatio equals to 1 only enable cpu
    
    bool cpu_enable = true;
    bool gpu_enable = true;

    vector<LoopInfo> infos;
    LoopInfo loopInfo;
    splitRatio = 0.1;

    timer.start();
    do {
        numIteration++;
        finished = true;
        h_finished = true;
        splitIndex = numEdges * splitRatio;
        d_numBlock = (numEdges - splitIndex + 1) / (d_numThreadsPerBlock * d_numEdgesPerThread) + 1;
        msgToDeviceIndex = 0;
        msgToHostIndex = 0;

        timer_gpu.start();
        timer_cpu.start();
        #pragma omp parallel //num_threads(8)
        {   
            int threadId = omp_get_thread_num();
            int h_numThreads = omp_get_num_threads();
            if (threadId == (h_numThreads - 1) && splitIndex < numEdges  && gpu_enable) {
                // Last thread will be used to launch gpu kernel 
                // if thread 0 is used to launch gpu kernel, the first block of 
                // data whose index begining from 0 will not be processed.
                gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
                // timer_host_to_device.start();
                // gpuErrorcheck(hipMemcpy(d_dist, dist, sizeof(uint) * numNodes, hipMemcpyHostToDevice));
                // timer_host_to_device.stop();
                gpuErrorcheck(hipMemcpy(d_msgToHostIndex, &msgToHostIndex, sizeof(uint), hipMemcpyHostToDevice));

                sssp_Hybrid_GPU_Kernel<<< d_numBlock, d_numThreadsPerBlock>>> (splitIndex,
                                                                        numEdges,
                                                                        d_numEdgesPerThread,
                                                                        d_dist,
                                                                        d_preNode,
                                                                        d_edgesSource,
                                                                        d_edgesEnd,
                                                                        d_edgesWeight,
                                                                        d_finished,
                                                                        d_msgToHostIndex,
                                                                        d_msgToHostNodeId,
                                                                        d_msgToHostDist);
                gpuErrorcheck(hipPeekAtLastError());
                gpuErrorcheck(hipDeviceSynchronize()); 
                gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
                // timer_device_to_host.start();
                // gpuErrorcheck(hipMemcpy(dist_copy, d_dist, sizeof(uint) * numNodes, hipMemcpyDeviceToHost));
                // timer_device_to_host.stop();

                gpuErrorcheck(hipMemcpy(&msgToHostIndex, d_msgToHostIndex, sizeof(uint), hipMemcpyDeviceToHost));
                gpuErrorcheck(hipMemcpy(msgToHostNodeId, d_msgToHostNodeId, sizeof(uint) * msgToHostIndex, hipMemcpyDeviceToHost));
                // printf("msgtohostindex: %d\n", msgToHostIndex);
                gpuErrorcheck(hipMemcpy(msgToHostDist, d_msgToHostDist, sizeof(uint) * msgToHostIndex, hipMemcpyDeviceToHost));

                timer_gpu.stop();
            } else if (cpu_enable) {
                // printf("Sub threads\n");
                int h_numEdgesPerThread = (splitIndex) / (h_numThreads - 1) + 1;
                
                sssp_Hybrid_CPU(threadId, 
                                splitIndex,
                                numEdges,
                                h_numEdgesPerThread,
                                dist,
                                preNode,
                                edgesSource,
                                edgesEnd,
                                edgesWeight,
                                &finished,
                                &msgToDeviceIndex,
                                msgToDeviceNodeId,
                                msgToDeviceDist);
                timer_cpu.stop();
            }
            
        }
        // printf("msgToDeviceIndex: %d \n", msgToDeviceIndex);
       
        finished = finished && h_finished;


        timer_cpu_message.start();
        timer_gpu_message.start();
        // Merge data
        #pragma omp parallel
        {
            int threadId = omp_get_thread_num();
            int h_numThreads = omp_get_num_threads();
            if (threadId == (h_numThreads - 1)) {
                int d_numMsg = msgToDeviceIndex;
                int d_numMsgPerThread = 8;
                d_numBlock = (d_numMsg) / (d_numThreadsPerBlock * d_numMsgPerThread) + 1;
                gpuErrorcheck(hipMemcpy(d_msgToDeviceNodeId, msgToDeviceNodeId, sizeof(uint) * d_numMsg, hipMemcpyHostToDevice));
                gpuErrorcheck(hipMemcpy(d_msgToDeviceDist, msgToDeviceDist, sizeof(uint) * d_numMsg, hipMemcpyHostToDevice));

                sssp_Hybrid_GPU_Process_Message<<< d_numBlock, d_numThreadsPerBlock >>> (d_numMsgPerThread,
                                                                                    d_numMsg,
                                                                                    d_dist,
                                                                                    d_preNode,
                                                                                    d_msgToDeviceNodeId,
                                                                                    d_msgToDeviceDist);
                gpuErrorcheck(hipPeekAtLastError());
                gpuErrorcheck(hipDeviceSynchronize());  
                timer_gpu_message.stop();
            } else if (threadId != (h_numThreads - 1)) {
                int h_numMsg = msgToHostIndex;
                int h_numMsgPerThread = (h_numMsg) / (h_numThreads - 1) + 1;
                sssp_Hybrid_Host_Process_Message(threadId,
                                                h_numMsg,
                                                h_numMsgPerThread,
                                                dist,
                                                msgToHostNodeId,
                                                msgToHostDist);

                timer_cpu_message.stop();
            }
        }

        // Load Balancing
        if (cpu_enable && gpu_enable) {
            float factor = (timer_cpu.elapsedTime() / timer_gpu.elapsedTime());
            if (factor > 1.1) {
                splitRatio = splitRatio - 0.05;
                if (splitRatio < 0) {
                    splitRatio = 0;
                    cpu_enable = false;
                }
    
            } else if (factor < 0.9) {
                splitRatio = splitRatio + 0.05;
                if (splitRatio > 1) {
                    splitRatio = 1;
                    gpu_enable = false;
                }
            }
            // printf("Copy dist from host to device : %f ms \n", timer_host_to_device.elapsedTime());
            // printf("Copy dist from device to host : %f ms \n", timer_device_to_host.elapsedTime());
            loopInfo.numIteration = numIteration;
            loopInfo.time_cpu = timer_cpu.elapsedTime() > 0 ? timer_cpu.elapsedTime() : 0;
            loopInfo.time_gpu = timer_gpu.elapsedTime() > 0 ? timer_gpu.elapsedTime() : 0;
            loopInfo.time_cpu_message = timer_cpu_message.elapsedTime() > 0 ? timer_cpu_message.elapsedTime() : 0;
            loopInfo.time_gpu_message = timer_gpu_message.elapsedTime() > 0 ? timer_gpu_message.elapsedTime() : 0;
            loopInfo.splitRatio = splitRatio;
            infos.push_back(loopInfo);
        } 
    } while(!finished);
    timer.stop();

    // printLoopInfoV2(infos);
    // printf("Process Done!\n");
    // printf("Number of Iteration: %d\n", numIteration);
    printf("The execution time of SSSP on Hybrid(CPU-GPU): %f ms\n", timer.elapsedTime());

    gpuErrorcheck(hipFree(d_dist));
    gpuErrorcheck(hipFree(d_preNode));
    gpuErrorcheck(hipFree(d_finished));
    gpuErrorcheck(hipFree(d_edgesSource));
    gpuErrorcheck(hipFree(d_edgesEnd));
    gpuErrorcheck(hipFree(d_edgesWeight));
    gpuErrorcheck(hipFree(d_msgToHostNodeId));
    gpuErrorcheck(hipFree(d_msgToHostDist));
    gpuErrorcheck(hipFree(d_msgToHostIndex));
    gpuErrorcheck(hipFree(d_msgToDeviceNodeId));
    gpuErrorcheck(hipFree(d_msgToDeviceDist));
    gpuErrorcheck(hipFree(d_msgToDeviceIndex));

    delete [] edgesSource;
    delete [] edgesEnd;
    delete [] edgesWeight;
    delete [] msgToHostNodeId;
    delete [] msgToHostDist;
    delete [] msgToDeviceNodeId;
    delete [] msgToDeviceDist;


    return dist;
}



int main(int argc, char **argv) {
    Timer timer_total, timer_load;
    timer_total.start();
    
    ArgumentParser args(argc, argv);

    timer_load.start();
    Graph graph(args.inputFilePath);
    //Graph graph("datasets/simpleGraph.txt");
    graph.readGraph();
    timer_load.stop();
    

    int sourceNode;

    if (args.hasSourceNode) {
        sourceNode = args.sourceNode;
    } else {
        // Use graph default source 
        sourceNode = graph.defaultSource;
    }


    uint *dist_hybrid;
    uint *dist_gpu = sssp_GPU(&graph, sourceNode);
    for (int i = 0; i < 100; i++) {
        dist_hybrid = sssp_Hybrid(&graph, sourceNode);
        compareResult(dist_hybrid, dist_gpu, graph.numNodes);
    }

    if (args.runOnCPU) {
        uint *dist_cpu = sssp_CPU_parallel(&graph, sourceNode);
        compareResult(dist_cpu, dist_hybrid, graph.numNodes);
    }

    timer_total.stop();
    printf("Total execution time: %f ms\n", timer_total.elapsedTime());
    printf("Graph loading execution time: %f ms\n", timer_load.elapsedTime());

    return 0;
}