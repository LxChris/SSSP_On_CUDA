#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "./utilities/timer.hpp"
#include "./utilities/graph.hpp"
#include "./utilities/gpu_error_check.cuh"

/*
Version 7. 128 Nodes Per Thread

*/

int numNodes;
int numEdges;

int* dist;
int* previousNode;
int ** graph;
bool* finished;
int* graph_static;

void init(Graph* graphData, int source) {
    numNodes = graphData->numNodes;
    graph = graphData->graph;

    int size = numNodes;

    dist = new int[size];
    previousNode = new int[size];
    finished = new bool[size];

    for (int i = 0; i < size; i++) {
        // set the distance to the source node
        dist[i] = graph[source][i];
        finished[i] = false;
        if (dist[i] == MAX_DIST) {
            previousNode[i] = -1;
        }
        else {
            previousNode[i] = source;
        }
    }
    dist[source] = 0;
}

void printShortestDistance(int source) {
    int diameter = 0;
    for (int i = 0; i < numNodes; i++) {
        if(dist[i] != MAX_DIST){
            if(dist[i] > diameter){
                diameter = dist[i];
            }
            // printf("Shortest distance from node: %d to source: %d: is: %d\n", i, source, dist[i]);
        }else{
            // printf("Shortest distance from node: %d to source: %d: is: INF\n", i, source);
        } 
    }
    printf("Maximum shortest distance : %d\n", diameter);
}

void printGraph(){
    printf("\n\nGraph");
    for (int i = 0; i < numNodes; i++){
        for (int j = 0; j < numNodes; j++){
            printf("%d ", graph[i][j]);
        }
        printf("\n");
    }
}

void printFinished(){
    printf("Finished array\n");
    for(int i = 0; i < numNodes; i++){
        printf("Node: %d, status: %d\n", i, finished[i]);
    }
    printf("\n");
}

void imcompletedAndConnectedNode(){
    int count = 0;
    for(int i = 0; i < numNodes; i++){
        if((!finished[i] && (dist[i] != MAX_DIST))){
            count++;
        }
    }
    printf("Imcompleted Nodes Number: %d\n", count);
}

void dijkstraOnCPU(int source) {
    Timer timer;
    int size = numNodes;
    int numIteration = 0;
    
    // Find the connected nodes to the source point
    // Set the source point
    dist[source] = 0;
    finished[source] = true;

    timer.start();
    for (int i = 0; i < size; i++) {
        int mindist = MAX_DIST;
        // U is the closet point to source, u is not finished yet
        int u = source;

        numIteration++;

        for (int j = 0; j < size; j++) {
            if ((j != u) && (!finished[j]) && dist[j] < mindist) {
                u = j;
                mindist = dist[j];
            }
        }
        // imcompletedAndConnectedNode();
        if (u == source){
            // Completed
            break;
        }

        finished[u] = true;

        for (int j = 0; j < size; j++) {
            if ((j != u) && (!finished[j]) && graph[u][j] < MAX_DIST) {
                // Find the shorter path
                if (dist[u] + graph[u][j] < dist[j]) {
                    // Update dist
                    dist[j] = dist[u] + graph[u][j];
                    // Update its previous point
                    previousNode[j] = u;
                }
            }
        }
    }
    printf("Number of Iteration Executed: %d\n", numIteration);
    printf("The execution time of SSSP on CPU: %d ms\n", timer.stop());
}

__global__ void dijkstraOnGPU_kernel1(int numNodes, 
                                        int sourceId,
                                        int partSize,
                                        int* graphData,
                                        bool* finished,
                                        int* dist,
                                        int* prev,
                                        int* closestNodeId,
                                        int* minimumDist,
                                        bool* completed) {
    // kernel 1 find the minimum distance of unfinshed node
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startNodeId = threadId * partSize;
    int endNodeId = (threadId + 1) * partSize;
    if(endNodeId > numNodes){
        endNodeId = numNodes;
    } 

    if(startNodeId > numNodes) return; 

    for(int nodeId = startNodeId; nodeId < endNodeId; nodeId++){
        if (!finished[nodeId] && dist[nodeId] < *minimumDist){
            *closestNodeId = nodeId;
            *minimumDist = dist[nodeId];
            *completed = false;
        }
    }

}

__global__ void dijkstraOnGPU_kernel2(int numNodes, 
                                        int sourceId,
                                        int partSize,
                                        int* graphData,
                                        bool* finished,
                                        int* dist,
                                        int* prev,
                                        int* closestNodeId,
                                        int minimumDist,
                                        int GRAPH_MAX_SIZE) {

    // Based on the minimum  node then update its connected node
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startNodeId = threadId * partSize;
    int endNodeId = (threadId + 1) * partSize;
    if(endNodeId > numNodes){
        endNodeId = numNodes;
    } 
    // int nodeId = threadId;

    if(startNodeId > numNodes) return;

    for (int nodeId = startNodeId; nodeId < endNodeId; nodeId++){
        // Version 2
        // Process the nodes, whose dist = minimumDist
        
         if ((!finished[nodeId]) && dist[nodeId] == minimumDist){
            finished[nodeId] = true;
            for (int connectedNodeId = 0; connectedNodeId < numNodes; connectedNodeId++){
                int index = nodeId * GRAPH_MAX_SIZE + connectedNodeId;
                if ((nodeId != connectedNodeId) && (graphData[index] < MAX_DIST)){
                    if (dist[nodeId] + graphData[index] < dist[connectedNodeId]){
                        finished[connectedNodeId] = false;
                        dist[connectedNodeId] = dist[nodeId] + graphData[index]; 
                    }
                }
            }
        }      
    }
}

void dijkstraOnGPU(int source){
    Timer timer;
    hipFree(0);
    // Define CPU vars
    int closestNodeId = 6;
    // Define GPU vars
    int* d_graph;   // 2D array is converted to 1-D, row = i / cols, col = i % cols;
    int* d_dist;
    int* d_prev;
    bool* d_finished;
    int* d_closestNodeId;
    int* d_minimumDist;
    bool* d_completed;

    gpuErrorcheck(hipMalloc((void **)&d_graph, GRAPH_MAX_SIZE * GRAPH_MAX_SIZE * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_dist, numNodes * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_prev, numNodes * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_finished, numNodes * sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_closestNodeId, sizeof(int)));
    gpuErrorcheck(hipMalloc(&d_completed, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_minimumDist, sizeof(int)));

    gpuErrorcheck(hipMemcpy(d_graph, graph[0], GRAPH_MAX_SIZE * GRAPH_MAX_SIZE * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_dist, dist, numNodes * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_prev, previousNode, numNodes * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_finished, finished, numNodes * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_closestNodeId, &closestNodeId, sizeof(int), hipMemcpyHostToDevice));

    bool completed = true;
    int minimumDist = MAX_DIST;
    int numIteration = 0;
   
    int numNodesPerPart = 2;
    int numThreadPerBlock = 64;
    int numBlock = (numNodes) / (numNodesPerPart * numThreadPerBlock) + 1;
    
    timer.start();
    do{
        numIteration++;
        completed = true;
        
        minimumDist = MAX_DIST;
        for (int nodeId = 0; nodeId < numNodes; nodeId++){
            // printf("process node %d\n", nodeId);
            if(!finished[nodeId] && dist[nodeId] < minimumDist){
                minimumDist = dist[nodeId];
                completed = false;
            }
        }

        dijkstraOnGPU_kernel2<<<numBlock, numThreadPerBlock>>>(numNodes,
            source,
            numNodesPerPart,
            d_graph,
            d_finished,
            d_dist,
            d_prev,
            d_closestNodeId,
            minimumDist,
            GRAPH_MAX_SIZE);

        gpuErrorcheck(hipMemcpy(finished, d_finished, sizeof(bool) * numNodes, hipMemcpyDeviceToHost));
        gpuErrorcheck(hipMemcpy(dist, d_dist, numNodes * sizeof(int), hipMemcpyDeviceToHost));

        gpuErrorcheck(hipPeekAtLastError());
        gpuErrorcheck(hipDeviceSynchronize());  
        
        
    }while(!completed);

    printf("Number of Iteration Executed: %d\n", numIteration);
    printf("The execution time of SSSP on GPU: %d ms\n", timer.stop());
    
    hipMemcpy(&closestNodeId, d_closestNodeId, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(dist, d_dist, numNodes * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_graph);
    hipFree(d_dist);
    hipFree(d_prev);
    hipFree(d_finished);
    hipFree(d_closestNodeId);
    hipFree(d_minimumDist);
    hipFree(d_completed);
}

int main() {

    // Graph graph1("datasets/simpleGragh2.txt");
    // Graph graph1("datasets/email-Eu-core-SIMPLE.txt");
    // Graph graph1("datasets/email-Eu-core.txt");
    Graph graph1("datasets/Wiki-Vote.txt");
    // Graph graph1("datasets/simpleGragh2.txt");
    // Graph graph1("datasets/CA-GrQc.txt");
     //Graph graph("datasets/testGraph.txt");
    graph1.readGraph();
    int sourceId = 30;

    init(&graph1, sourceId);   // source 0
        
    // Run SSSP on CPU
    dijkstraOnCPU(sourceId);
    printShortestDistance(sourceId);
    // printGraph();

    init(&graph1, sourceId);   // source 0
    // Run SSSP on GPU
    dijkstraOnGPU(sourceId);
    printShortestDistance(sourceId);
    // printGraph();
    return 0;
}