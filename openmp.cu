#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "./utilities/timer.hpp"
#include "./utilities/graph.hpp"
#include "./utilities/gpu_error_check.cuh"
#include "./utilities/global.hpp"
#include "./utilities/argument_parser.hpp"
#include <omp.h>


uint* sssp_CPU_parallel(Graph *graph, int source) {
    int numNodes = graph->numNodes;
    int numEdges = graph->numEdges;
    uint *dist = new uint[numNodes];
    uint *preNode = new uint[numNodes];
    bool *processed = new bool[numNodes];
    uint *edgesSource = new uint[numEdges];
    uint *edgesEnd = new uint[numEdges];
    uint *edgesWeight = new uint[numEdges];

    for (int i = 0; i < numNodes; i++) {
        dist[i] = MAX_DIST;
        preNode[i] = uint(-1);
        processed[i] = false;
    }

    for (int i = 0; i < numEdges;i ++) {
        Edge edge = graph->edges.at(i);
        edgesSource[i] = edge.source;
        edgesEnd[i] = edge.end;
        edgesWeight[i] = edge.weight;

        if (edge.source == source) {
            if (edge.weight < dist[edge.end]) {
                dist[edge.end] = edge.weight;
                preNode[edge.end] = source;
            }
        } else {
            // Case: edge.source != source
            continue;
        }
    }
    
    Timer timer;
    bool finished = false;
    uint numIteration = 0;

    dist[source] = 0;
    preNode[source] = 0;
    processed[source] = true;

    timer.start();
    while(!finished) {
        finished = true;
        numIteration++;
        
        #pragma omp parallel 
        {   
            // #pragma omp master 
            int threadId = omp_get_thread_num();
            int numThreads = omp_get_num_threads();
            int numEdgesPerThread = numEdges / numThreads + 1;
            int start = threadId * numEdgesPerThread;
            int end = (threadId + 1) * numEdgesPerThread;
            if (start > numEdges) {
                start = numEdges;
            }
            
            if (end > numEdges) {
                end = numEdges;
            }

            for (int i = start; i < end; i++) {
                uint source = edgesSource[i];
                uint end = edgesEnd[i];
                uint weight = edgesWeight[i];

                if (dist[source] + weight < dist[end]) {
                    // #pragma omp atomic
                    dist[end] = dist[source] + weight;
                    // #pragma omp atomic
                    preNode[end] = source;
                    finished = false;
                }
            }
        }
    }
    timer.stop();

    printf("Process Done!\n");
    printf("Number of Iteration: %d\n", numIteration);
    printf("The execution time of SSSP on CPU(OpenMP): %f ms\n", timer.elapsedTime());

    return dist;

}

__global__ void sssp_GPU_Kernel(int numEdges,
                                int numEdgesPerThread,
                                uint *dist,
                                uint *preNode,
                                uint *edgesSource,
                                uint *edgesEnd,
                                uint *edgesWeight,
                                bool *finished) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startId = threadId * numEdgesPerThread;

    if (startId >= numEdges) {
        return;
    }

    int endId = (threadId + 1) * numEdgesPerThread;
    if (endId >= numEdges) {
        endId = numEdges;
    }

    for (int nodeId = startId; nodeId < endId; nodeId++) {
        uint source = edgesSource[nodeId];
        uint end = edgesEnd[nodeId];
        uint weight = edgesWeight[nodeId];

        if (dist[source] + weight < dist[end]) {
        atomicMin(&dist[end], dist[source] + weight);
        // dist[end] = dist[source] + weight;
        preNode[end] = source;
        *finished = false;
        }
    }

}

uint* sssp_GPU(Graph *graph, int source) {
    int numNodes = graph->numNodes;
    int numEdges = graph->numEdges;
    uint *dist = new uint[numNodes];
    uint *preNode = new uint[numNodes];
    uint *edgesSource = new uint[numEdges];
    uint *edgesEnd = new uint[numEdges];
    uint *edgesWeight = new uint[numEdges];

    for (int i = 0; i < numNodes; i++) {
        dist[i] = MAX_DIST;
        preNode[i] = uint(-1);
    }


    for (int i = 0; i < numEdges; i++) {
        Edge edge = graph->edges.at(i);

        // Transfer the vector to the following three arrays
        edgesSource[i] = edge.source;
        edgesEnd[i] = edge.end;
        edgesWeight[i] = edge.weight;

        if (edge.source == source){
            if (edge.weight < dist[edge.end]){
                dist[edge.end] = edge.weight;
                preNode[edge.end] = source;
            }
        } else {
        // Case: edge.source != source
        continue;
        }
    }

    dist[source] = 0;
    preNode[source] = 0;


    uint *d_dist;
    uint *d_preNode;
    bool *d_finished;
    uint *d_edgesSource;
    uint *d_edgesEnd;
    uint *d_edgesWeight;

    gpuErrorcheck(hipMalloc(&d_dist, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_preNode, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_edgesSource, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesEnd, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesWeight, numEdges * sizeof(uint)));

    gpuErrorcheck(hipMemcpy(d_dist, dist, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_preNode, preNode, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesSource, edgesSource, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesEnd, edgesEnd, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesWeight, edgesWeight, numEdges * sizeof(uint), hipMemcpyHostToDevice));

    Timer timer;
    int numIteration = 0;
    int numEdgesPerThread = 8;
    int numThreadsPerBlock = 512;
    int numBlock = (numEdges) / (numThreadsPerBlock * numEdgesPerThread) + 1;
    bool finished = true;

    timer.start();
    do {
        numIteration++;
        finished = true;

        gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

        // TO-DO PARALLEL
        sssp_GPU_Kernel<<< numBlock, numThreadsPerBlock >>> (numEdges,
                                        numEdgesPerThread,
                                        d_dist,
                                        d_preNode,
                                        d_edgesSource,
                                        d_edgesEnd,
                                        d_edgesWeight,
                                        d_finished);

        gpuErrorcheck(hipPeekAtLastError());
        gpuErrorcheck(hipDeviceSynchronize()); 
        gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
    } while(!finished);
    timer.stop();


    printf("Process Done!\n");
    printf("Number of Iteration: %d\n", numIteration);
    printf("The execution time of SSSP on GPU: %f ms\n", timer.elapsedTime());

    gpuErrorcheck(hipMemcpy(dist, d_dist, numNodes * sizeof(uint), hipMemcpyDeviceToHost));

    gpuErrorcheck(hipFree(d_dist));
    gpuErrorcheck(hipFree(d_preNode));
    gpuErrorcheck(hipFree(d_finished));
    gpuErrorcheck(hipFree(d_edgesSource));
    gpuErrorcheck(hipFree(d_edgesEnd));
    gpuErrorcheck(hipFree(d_edgesWeight));

    return dist;
}


__global__ void sssp_GPU_Hybrid_Kernel(int splitIndex,
                                int numEdges,
                                int numEdgesPerThread,
                                uint *dist,
                                uint *preNode,
                                uint *edgesSource,
                                uint *edgesEnd,
                                uint *edgesWeight,
                                bool *finished) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int startId = splitIndex + threadId * numEdgesPerThread;
    if (startId >= numEdges) {
        return;
    }

    int endId = splitIndex + (threadId + 1) * numEdgesPerThread;
    if (endId >= numEdges) {
        endId = numEdges;
    }

    // printf("GPU: process edged from: %d to %d \n", startId, endId);
    for (int nodeId = startId; nodeId < endId; nodeId++) {
        uint source = edgesSource[nodeId];
        uint end = edgesEnd[nodeId];
        uint weight = edgesWeight[nodeId];
        
        if (dist[source] + weight < dist[end]) {
            atomicMin(&dist[end], dist[source] + weight);
            preNode[end] = source;
            *finished = false;
        }
    }
}

uint* sssp_Hybrid(Graph *graph, int source) {
    int numNodes = graph->numNodes;
    int numEdges = graph->numEdges;
    uint *dist = new uint[numNodes];
    uint *preNode = new uint[numNodes];
    uint *edgesSource = new uint[numEdges];
    uint *edgesEnd = new uint[numEdges];
    uint *edgesWeight = new uint[numEdges];
    uint *dist_copy = new uint[numNodes];

    for (int i = 0; i < numNodes; i++) {
        dist[i] = MAX_DIST;
        preNode[i] = uint(-1);
    }


    for (int i = 0; i < numEdges; i++) {
        Edge edge = graph->edges.at(i);
        
        // Transfer the vector to the following three arrays
        edgesSource[i] = edge.source;
        edgesEnd[i] = edge.end;
        edgesWeight[i] = edge.weight;

        if (edge.source == source){
            if (edge.weight < dist[edge.end]){
                dist[edge.end] = edge.weight;
                preNode[edge.end] = source;
            }
        } else {
            // Case: edge.source != source
            continue;
        }
    }

    dist[source] = 0;
    preNode[source] = 0;

    uint *d_dist;
    uint *d_preNode;
    bool *d_finished;
    uint *d_edgesSource;
    uint *d_edgesEnd;
    uint *d_edgesWeight;

    gpuErrorcheck(hipMalloc(&d_dist, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_preNode, numNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_edgesSource, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesEnd, numEdges * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_edgesWeight, numEdges * sizeof(uint)));

    gpuErrorcheck(hipMemcpy(d_dist, dist, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_preNode, preNode, numNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesSource, edgesSource, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesEnd, edgesEnd, numEdges * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgesWeight, edgesWeight, numEdges * sizeof(uint), hipMemcpyHostToDevice));

    // Copy from gpu memory
    memcpy(dist_copy, dist, numNodes * sizeof(uint));

    Timer timer;
    int numIteration = 0;
    bool finished = false;
    bool h_finished = false;
    
    
    float splitRatio; // cpu_data_size / whole_data_size

    // Automatic select a prior value of spritRatio based on experience
    if (numEdges < 300000) {
        splitRatio = 0.95;
    } else if (numEdges < 800000) {
        splitRatio = 0.7;
    } else {
        splitRatio = 0.5;
    }

    /*
    CPU process edges from 0 to splitIndex   
        number of edges: splitIndex
    GPU process edges from splitIndex to numEdges 
        number of edges: numEdges - splitIndex + 1
    */
    int splitIndex = numEdges * splitRatio;
    int d_numEdgesPerThread = 8;
    int d_numThreadsPerBlock = 512;
    int d_numBlock = (numEdges - splitIndex + 1) / (d_numThreadsPerBlock * d_numEdgesPerThread) + 1;
    
    Timer timer_cpu, timer_gpu;
    Timer timer_host_to_device;
    Timer timer_device_to_host;

    // Default: enable cpu and gpu 
    // Once splitRatio equals to 0 only enable gpu
    // Once splitRatio equals to 1 only enable cpu
    
    bool cpu_enable = true;
    bool gpu_enable = true;

    vector<LoopInfo> infos;
    LoopInfo loopInfo;

    Timer timer_merge;
    timer.start();
    do {
        numIteration++;
        finished = true;
        h_finished = true;
        splitIndex = numEdges * splitRatio;
        d_numBlock = (numEdges - splitIndex + 1) / (d_numThreadsPerBlock * d_numEdgesPerThread) + 1;
        
        timer_gpu.start();
        timer_cpu.start();
        #pragma omp parallel //num_threads(8)
        {   
            int threadId = omp_get_thread_num();
            int h_numThreads = omp_get_num_threads();
            if (threadId == h_numThreads - 1 && splitIndex < numEdges  && gpu_enable) {
                // Last thread will be used to launch gpu kernel 
                // if thread 0 is used to launch gpu kernel, the first block of 
                // data whose index begining from 0 will not be processed.
                gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
                // timer_host_to_device.start();
                gpuErrorcheck(hipMemcpy(d_dist, dist, sizeof(uint) * numNodes, hipMemcpyHostToDevice));
                // timer_host_to_device.stop();
                sssp_GPU_Hybrid_Kernel<<< d_numBlock, d_numThreadsPerBlock>>> (splitIndex,
                                                                        numEdges,
                                                                        d_numEdgesPerThread,
                                                                        d_dist,
                                                                        d_preNode,
                                                                        d_edgesSource,
                                                                        d_edgesEnd,
                                                                        d_edgesWeight,
                                                                        d_finished);
                gpuErrorcheck(hipPeekAtLastError());
                gpuErrorcheck(hipDeviceSynchronize()); 
                gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
                // timer_device_to_host.start();
                gpuErrorcheck(hipMemcpy(dist_copy, d_dist, sizeof(uint) * numNodes, hipMemcpyDeviceToHost));
                // timer_device_to_host.stop();
                timer_gpu.stop();
            } else if (cpu_enable) {
                // printf("Sub threads\n");
                int h_numEdgesPerThread = (splitIndex) / (h_numThreads - 1) + 1;
                int start = threadId * h_numEdgesPerThread;
                int end = (threadId + 1) * h_numEdgesPerThread;
                if (start > splitIndex) {
                    start = splitIndex;
                }
                if (end > splitIndex) {
                    end = splitIndex;
                }

                // cout << "Processs node: from " << start << " to: " << end << endl;
                // printf("Process node from: %d to : %d\n", start, end);
                for (int i = start; i < end; i++) {
                    uint source = edgesSource[i];
                    uint end = edgesEnd[i];
                    uint weight = edgesWeight[i];
                    
                    if (dist[source] + weight < dist[end]) {
                        dist[end] = dist[source] + weight;
                        preNode[end] = source;
                        h_finished = false;
                    }
                }
                timer_cpu.stop();
            }
        }
        
       
        finished = finished && h_finished;
        timer_merge.start();
        #pragma omp parallel //num_threads(8)
        {
            int threadId = omp_get_thread_num();
            int h_numThreads = omp_get_num_threads();
            int h_numNodesPerThread = (numNodes) / (h_numThreads) + 1;
            if (!finished) {
                // Merge
                int startIdx = threadId * h_numNodesPerThread;
                int endIdx = (threadId + 1) * h_numNodesPerThread;
                if (startIdx > numNodes) {
                    startIdx = numNodes;
                }
                if (endIdx > numNodes) {
                    endIdx = numNodes;
                }
                for (int i = startIdx; i < endIdx; i++) {
                    if (dist[i] > dist_copy[i]) {
                        dist[i] = dist_copy[i];
                    }
                }
            }
        }
        timer_merge.stop();
        // Load Balancing

        if (cpu_enable && gpu_enable) {
            float factor = (timer_cpu.elapsedTime() / timer_gpu.elapsedTime());
            if (factor > 1.1) {
                splitRatio = splitRatio - 0.05;
                if (splitRatio < 0) {
                    splitRatio = 0;
                    cpu_enable = false;
                }
    
            } else if (factor < 0.9) {
                splitRatio = splitRatio + 0.05;
                if (splitRatio > 1) {
                    splitRatio = 1;
                    gpu_enable = false;
                }
            }

            // printf("No. itr: %d , updated splitRatio: %f, factor: %f\n", numIteration, splitRatio, factor);

            // printf("CPU PART TIME: %f\n", timer_cpu.elapsedTime());
            // printf("GPU PART TIME: %f\n", timer_gpu.elapsedTime());
            // printf("Copy dist from host to device : %f ms \n", timer_host_to_device.elapsedTime());
            // printf("Copy dist from device to host : %f ms \n", timer_device_to_host.elapsedTime());
            loopInfo.numIteration = numIteration;
            loopInfo.time_cpu = timer_cpu.elapsedTime() > 0 ? timer_cpu.elapsedTime() : 0;
            loopInfo.time_gpu = timer_gpu.elapsedTime() > 0 ? timer_gpu.elapsedTime() : 0;
            loopInfo.time_dist_merge = timer_merge.elapsedTime();
            loopInfo.splitRatio = splitRatio;
            infos.push_back(loopInfo);
        } 
    } while(!finished);
    timer.stop();

    printLoopInfoV1(infos);
    printf("Process Done!\n");
    printf("Number of Iteration: %d\n", numIteration);
    printf("The execution time of SSSP on Hybrid(CPU-GPU): %f ms\n", timer.elapsedTime());

    gpuErrorcheck(hipFree(d_dist));
    gpuErrorcheck(hipFree(d_preNode));
    gpuErrorcheck(hipFree(d_finished));
    gpuErrorcheck(hipFree(d_edgesSource));
    gpuErrorcheck(hipFree(d_edgesEnd));
    gpuErrorcheck(hipFree(d_edgesWeight));


    return dist;
}



int main(int argc, char **argv) {
    Timer timer_total, timer_load;
    timer_total.start();
    
    ArgumentParser args(argc, argv);

    timer_load.start();
    Graph graph(args.inputFilePath);
    //Graph graph("datasets/simpleGraph.txt");
    graph.readGraph();
    timer_load.stop();
    

    int sourceNode;

    if (args.hasSourceNode) {
        sourceNode = args.sourceNode;
    } else {
        // Use graph default source 
        sourceNode = graph.defaultSource;
    }


    uint *dist_hybrid = sssp_Hybrid(&graph, sourceNode);
    uint *dist_gpu = sssp_GPU(&graph, sourceNode);

    compareResult(dist_hybrid, dist_gpu, graph.numNodes);

    if (args.runOnCPU) {
        uint *dist_cpu = sssp_CPU_parallel(&graph, sourceNode);
        compareResult(dist_cpu, dist_hybrid, graph.numNodes);
    }

    timer_total.stop();
    printf("Total execution time: %f ms\n", timer_total.elapsedTime());
    printf("Graph loading execution time: %f ms\n", timer_load.elapsedTime());

    return 0;
}